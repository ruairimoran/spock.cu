#include "hip/hip_runtime.h"
#include "../include/gpu.cuh"


/**
 * Definitions for CUDA kernels 
*/


/*
 * General
*/

__host__ __device__ size_t getIdxMat(size_t node, size_t row, size_t col, size_t rows, size_t cols = 0) {
    if (cols == 0) cols = rows;
    return (node * rows * cols) + (col * rows + row);
}

__global__ void d_setMatToId(real_t *mat, size_t numRows, size_t node = 0) {
    if (blockIdx.x == threadIdx.x) {
        size_t idx = getIdxMat(node, blockIdx.x, threadIdx.x, numRows);
        mat[idx] = 1.0;
    }
}


/*
 * Cache methods
*/


/*
 * Constraints methods
*/

__global__ void d_projectRectangle(size_t dimension, real_t *vec, real_t *lowerBound, real_t *upperBound) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < dimension) {
        if (vec[i] < lowerBound[i]) vec[i] = lowerBound[i];
        if (vec[i] > upperBound[i]) vec[i] = upperBound[i];
    }
}


/*
 * Risk methods
*/

__global__ void d_avarVecAddB(real_t *vec, size_t node, size_t *numCh, size_t *chFrom, real_t *probs) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < numCh[node]) vec[i] += probs[chFrom[node] + i];
}


/*
 * Cone methods
*/

__global__ void d_maxWithZero(real_t *vec, size_t n) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) vec[i] = max(0., vec[i]);
}

__global__ void d_setToZero(real_t *vec, size_t n) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) vec[i] = 0.;
}

__global__ void d_projectOnSoc(real_t *vec, size_t n, real_t nrm, real_t scaling) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n - 1) vec[i] *= scaling;
    if (i == n - 1) vec[i] = scaling * nrm;
}


/*
 * ScenarioTree methods
*/

/**
 * Computing conditional probability of each tree node
 *
 * @param[in] anc device ptr to ancestor of node at index
 * @param[in] prob device ptr to probability of visiting node at index
 * @param[in] numNodes total number of nodes
 * @param[out] condProb device ptr to conditional probability of visiting node at index, given ancestor node visited
 */
__global__ void d_populateProbabilities(size_t *anc, real_t *prob, size_t numNodes, real_t *condProb) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i == 0) {
        condProb[i] = 1.0;
    } else if (i < numNodes) {
        condProb[i] = prob[i] / prob[anc[i]];
    }
}

/**
 * Computing number of children of each tree node
 *
 * @param[in] from device ptr to first child of node at index
 * @param[in] to device ptr to last child of node at index
 * @param[in] numNonleafNodes total number of nonleaf nodes
 * @param[out] numChildren device ptr to number of children of node at index
 */
__global__ void d_populateChildren(size_t *from, size_t *to, size_t numNonleafNodes, size_t *numChildren) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < numNonleafNodes) numChildren[i] = to[i] - from[i] + 1;
}

/**
 * Populating stagesFrom and stagesTo
 *
 * @param[in] stages device ptr to stage of node at index
 * @param[in] numStages total number of stages
 * @param[out] nodeFrom device ptr to first node of stage at index
 * @param[out] nodeTo device ptr to last node of stage at index
 */
__global__ void d_populateStages(size_t *stages, size_t numStages, size_t numNodes, size_t *nodeFrom, size_t *nodeTo) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < numStages) {
        for (size_t j = 0; j < numNodes; j++) {
            if (stages[j] == i) {
                nodeFrom[i] = j;
                break;
            }
        }
        for (size_t j = numNodes - 1;; j--) {
            if (stages[j] == i) {
                nodeTo[i] = j;
                break;
            }
        }
    }
}
