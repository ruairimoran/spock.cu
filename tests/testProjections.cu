#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include <numeric>
#include "../src/projections.cuh"
#include "../src/tree.cuh"


class ProjectionsTest : public testing::Test {

protected:
    /* Prepare some host and device data */
    size_t m_n = 520;
    DTensor<DEFAULT_FPX> m_d_data = DTensor<DEFAULT_FPX>(m_n, 1, 1, true);
    std::vector<DEFAULT_FPX> m_data = std::vector<DEFAULT_FPX>(m_n);
    std::vector<DEFAULT_FPX> m_socA = std::vector<DEFAULT_FPX>(m_n);
    std::vector<DEFAULT_FPX> m_socB = std::vector<DEFAULT_FPX>(m_n);
    std::vector<DEFAULT_FPX> m_socC = std::vector<DEFAULT_FPX>(m_n);
    std::vector<DEFAULT_FPX> m_test = std::vector<DEFAULT_FPX>(m_n);
    std::vector<DEFAULT_FPX> m_zero = std::vector<DEFAULT_FPX>(m_n);
    DTensor<DEFAULT_FPX> d_singleProjectSize = DTensor<DEFAULT_FPX>(m_d_data.numRows());
    SocProjection<DEFAULT_FPX> socProj = SocProjection<DEFAULT_FPX>(d_singleProjectSize);

    ProjectionsTest() {
        /* For testing i1 */
        for (size_t i = 0; i < m_n - 1; i++) { m_socA[i] = 0.; }
        m_socA[m_n - 1] = 1.;
        /* For testing i2 */
        for (size_t i = 0; i < m_n - 1; i++) { m_socB[i] = 0.; }
        m_socB[m_n - 1] = -1.;
        /* For testing i3 */
        for (size_t i = 0; i < m_n - 1; i++) { m_socC[i] = 1.; }
        m_socC[m_n - 1] = 0.;
    }

    virtual ~ProjectionsTest() {}
};

TEMPLATE_WITH_TYPE_T
void socProjectSerial(size_t dim, std::vector<T> &vec) {
    std::vector<T> vecFirstPart(vec.begin(), vec.end() - 1);
    std::vector<T> squares(dim - 1);
    T sum = 0;
    for (size_t i = 0; i < dim - 2; i++) {
        T temp = vecFirstPart[i];
        squares[i] = temp * temp;
        sum += squares[i];
    }
    T nrm = sqrt(sum);
    float vecLastElement = vec[dim - 1];
    if (nrm <= vecLastElement) {
        return;  // Do nothing!
    } else if (nrm <= -vecLastElement) {
        for (size_t i = 0; i < dim; i++) { vec[i] = 0.; }
    } else {
        T scaling = (nrm + vecLastElement) / (2. * nrm);
        for (size_t i = 0; i < dim - 1; i++) { vec[i] *= scaling; }
        vec[dim - 1] = scaling * nrm;
    }
}

TEMPLATE_WITH_TYPE_T
void testSocElse(std::vector<T> testVec) {
    T last = testVec[testVec.size() - 1];
    testVec.pop_back();
    T nrm = std::sqrt(std::inner_product(testVec.begin(), testVec.end(), testVec.begin(), 0.));
    EXPECT_TRUE(nrm <= last);
}

TEST_F(ProjectionsTest, SocProjI3) {
    m_d_data.upload(m_socC);
    socProj.project(m_d_data);
    m_d_data.download(m_test);
    testSocElse(m_test);
}

TEST_F(ProjectionsTest, SocProjI2) {
    m_d_data.upload(m_socB);
    socProj.project(m_d_data);
    m_d_data.download(m_test);
    EXPECT_EQ(m_test, m_zero);
}

TEST_F(ProjectionsTest, SocProjI1) {
    m_d_data.upload(m_socA);
    socProj.project(m_d_data);
    m_d_data.download(m_test);
    EXPECT_EQ(m_test, m_socA);
}

TEST_F(ProjectionsTest, SocProjI321) {
    /* If the norms of SocProjection are not reset to zeros before each projection,
     * this test will fail.
     */
    m_d_data.upload(m_socC);
    socProj.project(m_d_data);
    m_d_data.download(m_test);
    testSocElse(m_test);
    m_d_data.upload(m_socB);
    socProj.project(m_d_data);
    m_d_data.download(m_test);
    EXPECT_EQ(m_test, m_zero);
    m_d_data.upload(m_socA);
    socProj.project(m_d_data);
    m_d_data.download(m_test);
    EXPECT_EQ(m_test, m_socA);
}

TEST_F(ProjectionsTest, CartesianCone) {
    size_t coneDim = 5;
    size_t numCones = 3;
    std::vector<DEFAULT_FPX> socs = {1., 2., 3., 4., 0.5,
                                     5., 6., 7., 8., -200,
                                     9., -10., 11., -12., 100};
    DTensor<DEFAULT_FPX> d_socs(socs, coneDim, numCones);
    SocProjection multiSocProj(d_socs);
    multiSocProj.project(d_socs);
    std::vector<DEFAULT_FPX> test(coneDim, numCones);
    d_socs.download(test);
    std::vector<DEFAULT_FPX> expected = {0.5456435464587639, 1.0912870929175278, 1.6369306393762917, 2.1825741858350556,
                                         2.988612787525831,
                                         0., 0., 0., 0., 0.,
                                         9., -10., 11., -12., 100.};
    multiSocProj.project(d_socs);
    EXPECT_EQ(test, expected);
}

TEST_F(ProjectionsTest, Serial) {
    size_t coneDim = 5;
    size_t numCones = 3;
    std::vector<DEFAULT_FPX> socs = {1., 2., 3., 4., 0.5,
                                     5., 6., 7., 8., -200,
                                     9., -10., 11., -12., 100};
    std::vector<std::vector<DEFAULT_FPX>> split(numCones);
    for (size_t i = 0; i < numCones; i++) {
        split[i] = std::vector<DEFAULT_FPX>(socs.begin() + coneDim * i,
                                            socs.begin() + coneDim * (i + 1));
    }
    std::vector<std::vector<DEFAULT_FPX>> expected(numCones);
    expected[0] = {0.56681531047810607, 1.1336306209562121, 1.7004459314343183, 2.2672612419124243, 2.1208286933869704};
    expected[1] = {0., 0., 0., 0., 0.};
    expected[2] = {9., -10., 11., -12., 100.};
    for (size_t i = 0; i < numCones; i++) { socProjectSerial(coneDim, split[i]); }
    for (size_t i = 0; i < numCones; i++) { EXPECT_EQ(split[i], expected[i]); }
}


