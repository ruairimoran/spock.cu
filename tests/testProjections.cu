#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include <numeric>
#include "../src/projections.cuh"
#include "../src/tree.cuh"


class ProjectionsTest : public testing::Test {

protected:
    /** Prepare some host and device data */
    size_t m_n = 64;
    size_t m_numConeTypes = 4;
    DTensor<DEFAULT_FPX> m_d_data = DTensor<DEFAULT_FPX>(m_n);
    DTensor<DEFAULT_FPX> m_d_dataCart = DTensor<DEFAULT_FPX>(m_n * m_numConeTypes);
    std::vector<DEFAULT_FPX> m_hostData = std::vector<DEFAULT_FPX>(m_n);
    std::vector<DEFAULT_FPX> m_hostTest = std::vector<DEFAULT_FPX>(m_n);
    std::vector<DEFAULT_FPX> m_hostZero = std::vector<DEFAULT_FPX>(m_n);
    std::vector<DEFAULT_FPX> m_hostSocA = std::vector<DEFAULT_FPX>(m_n);
    std::vector<DEFAULT_FPX> m_hostSocB = std::vector<DEFAULT_FPX>(m_n);
    std::vector<DEFAULT_FPX> m_hostSocC = std::vector<DEFAULT_FPX>(m_n);
    std::vector<DEFAULT_FPX> m_hostCart = std::vector<DEFAULT_FPX>(m_n * m_numConeTypes);
    std::vector<DEFAULT_FPX> m_testCart;

    ProjectionsTest() {
        /** Positive and negative values in m_hostData */
        for (size_t i = 0; i < m_n; i = i + 2) { m_hostData[i] = -2. * (i + 1.); }
        for (size_t i = 1; i < m_n; i = i + 2) { m_hostData[i] = 2. * (i + 1.); }
        m_d_data.upload(m_hostData);  ///< Main vector for projection testing
        /** Zeroes in m_hostZero */
        for (size_t i = 0; i < m_n; i++) { m_hostZero[i] = 0.; }
        /** For testing `if` projection of SOC */
        for (size_t i = 0; i < m_n - 1; i++) { m_hostSocA[i] = 0.; }
        m_hostSocA[m_n - 1] = 1.;
        /** For testing `else if` projection of SOC */
        for (size_t i = 0; i < m_n - 1; i++) { m_hostSocB[i] = 0.; }
        m_hostSocB[m_n - 1] = -1.;
        /** For testing `else` projection of SOC */
        for (size_t i = 0; i < m_n - 1; i++) { m_hostSocC[i] = 1.; }
        m_hostSocC[m_n - 1] = 0.;
        /** For testing Cartesian cone of all types */
        for (size_t i = 0; i < m_n * m_numConeTypes; i++) {
            if (m_n * 0 <= i && i < m_n * 1) m_hostCart[i] = m_hostData[i % m_n];  ///< For projecting Cartesian::Univ
            if (m_n * 1 <= i && i < m_n * 2) m_hostCart[i] = m_hostData[i % m_n];  ///< For projecting Cartesian::Zero
            if (m_n * 2 <= i && i < m_n * 3) m_hostCart[i] = m_hostData[i % m_n];  ///< For projecting Cartesian::NnOC
            if (m_n * 3 <= i && i < m_n * 4) m_hostCart[i] = m_hostSocC[i % m_n];  ///< For projecting Cartesian::SOC
        }
    }

    virtual ~ProjectionsTest() {}
};

TEMPLATE_WITH_TYPE_T
void testNnocProjection(std::vector<T> testVec) {
    for (size_t i=0; i<testVec.size(); i++) { EXPECT_TRUE(testVec[i] >= 0.); }
}

TEMPLATE_WITH_TYPE_T
void testSocElse(std::vector<T> testVec) {
    T last = testVec[testVec.size() - 1];
    testVec.pop_back();
    T nrm = std::sqrt(std::inner_product(testVec.begin(), testVec.end(), testVec.begin(), 0.));
    EXPECT_TRUE(nrm <= last);
}

TEST_F(ProjectionsTest, SecondOrderConeCone) {
//    SecondOrderCone myCone(m_n);
//    /** Testing `if` projection of SOC */
//    m_d_data.upload(m_hostSocA);
//    myCone.project(m_d_data);
//    m_d_data.download(m_hostTest);
//    EXPECT_TRUE((m_hostTest == m_hostSocA));
//    /** Testing `else if` projection of SOC */
//    m_d_data.upload(m_hostSocB);
//    myCone.project(m_d_data);
//    m_d_data.download(m_hostTest);
//    EXPECT_TRUE((m_hostTest == m_hostZero));
//    /** Testing `else` projection of SOC */
//    m_d_data.upload(m_hostSocC);
//    myCone.project(m_d_data);
//    m_d_data.download(m_hostTest);
//    testSocElse(m_hostTest);
}

TEST_F(ProjectionsTest, CartesianCone) {
//    // for (size_t i=0; i<m_n*m_numConeTypes; i++) { std::cerr << m_hostCart[i] << " "; }  ///< For debugging
//    m_d_dataCart.upload(m_hostCart);
//    UniverseCone myUniv(m_n);
//    ZeroCone myZero(m_n);
//    NonnegativeOrthantCone myNnoc(m_n);
//    SecondOrderCone mySoc(m_n);
//    Cartesian myCone;
//    myCone.addCone(myUniv);
//    myCone.addCone(myZero);
//    myCone.addCone(myNnoc);
//    myCone.addCone(mySoc);
//    myCone.project(m_d_dataCart);
//    m_d_dataCart.download(m_hostCart);
//    /** Test Universe cone */
//    size_t index = 0;
//    m_testCart = std::vector<DEFAULT_FPX>(m_hostCart.begin() + index, m_hostCart.begin() + index + m_n);
//    EXPECT_TRUE((m_testCart == m_hostData));
//    // for (size_t i=0; i<m_n; i++) { std::cerr << m_testCart[i] << " "; }  ///< For debugging
//    /** Test Zero cone */
//    index += m_n;
//    m_testCart = std::vector<DEFAULT_FPX>(m_hostCart.begin() + index, m_hostCart.begin() + index + m_n);
//    EXPECT_TRUE((m_testCart == m_hostZero));
//    // for (size_t i=0; i<m_n; i++) { std::cerr << m_testCart[i] << " "; }  ///< For debugging
//    /** Test NnOC cone */
//    index += m_n;
//    m_testCart = std::vector<DEFAULT_FPX>(m_hostCart.begin() + index, m_hostCart.begin() + index + m_n);
//    testNnocProjection(m_testCart);
//    // for (size_t i=0; i<m_n; i++) { std::cerr << m_testCart[i] << " "; }  ///< For debugging
//    /** Test SOC cone */
//    index += m_n;
//    m_testCart = std::vector<DEFAULT_FPX>(m_hostCart.begin() + index, m_hostCart.begin() + index + m_n);
//    testSocElse(m_testCart);
//    // for (size_t i=0; i<m_n; i++) { std::cerr << m_testCart[i] << " "; }  ///< For debugging
}


